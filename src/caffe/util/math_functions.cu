#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <stdio.h>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = alpha;
	}
}

template <>
void caffe_gpu_set(const int N, const float alpha, float* Y) {
	if (alpha == 0) {
		CUDA_CHECK(hipMemset(Y, 0, sizeof(float) * N));
		return;
	}
	// NOLINT_NEXT_LINE(whitespace/operators)
	set_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, alpha, Y);
}

template <>
void caffe_gpu_set(const int N, const double alpha, double* Y) {
	if (alpha == 0) {
		CUDA_CHECK(hipMemset(Y, 0, sizeof(double) * N));
		return;
	}
	// NOLINT_NEXT_LINE(whitespace/operators)
	set_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, alpha, Y);
}

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] += alpha;
	}
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, alpha, Y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
		const Dtype* b, Dtype* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = a[index] * b[index];
	}
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
		const float* b, float* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
		const double* b, double* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
		const Dtype* b, Dtype* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = a[index] / b[index];
	}
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
		const float* b, float* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
		const double* b, double* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, b, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
		const Dtype alpha, Dtype* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = pow(a[index], alpha);
	}
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
		const float alpha, float* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
		const double alpha, double* y) {
	// NOLINT_NEXT_LINE(whitespace/operators)
	powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
			N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
		- (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(fabs, y[index] = fabs(x[index]));

__global__ void popc_kernel(const int n, const float* a,
		const float* b, uint8_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = __popc(static_cast<uint32_t>(a[index]) ^
				static_cast<uint32_t>(b[index]));
	}
}

__global__ void popcll_kernel(const int n, const double* a,
		const double* b, uint8_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
				static_cast<uint64_t>(b[index]));
	}
}

template <>
uint32_t caffe_gpu_hamming_distance<float>(const int n, const float* x,
		const float* y) {
	// TODO: Fix caffe_gpu_hamming_distance (see failing unit test
	// TestHammingDistanceGPU in test_math_functions.cpp).
	NOT_IMPLEMENTED;
	thrust::device_vector<uint8_t> popcounts(n);
	// NOLINT_NEXT_LINE(whitespace/operators)
	popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
			n, x, y, thrust::raw_pointer_cast(popcounts.data()));
	return thrust::reduce(popcounts.begin(), popcounts.end(),
			(uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double>(const int n, const double* x,
		const double* y) {
	// TODO: Fix caffe_gpu_hamming_distance (see failing unit test
	// TestHammingDistanceGPU in test_math_functions.cpp).
	NOT_IMPLEMENTED;
	thrust::device_vector<uint8_t> popcounts(n);
	// NOLINT_NEXT_LINE(whitespace/operators)
	popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
			n, x, y, thrust::raw_pointer_cast(popcounts.data()));
	return thrust::reduce(popcounts.begin(), popcounts.end(),
			/* NOLINT_NEXT_LINE(build/include_what_you_use) */
			(uint32_t) 0, thrust::plus<uint32_t>());
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
	CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
		float* r) {
	CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
	const float range = b - a;
	if (range != static_cast<float>(1)) {
		caffe_gpu_scal(n, range, r);
	}
	if (a != static_cast<float>(0)) {
		caffe_gpu_add_scalar(n, a, r);
	}
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
		double* r) {
	CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
	const double range = b - a;
	if (range != static_cast<double>(1)) {
		caffe_gpu_scal(n, range, r);
	}
	if (a != static_cast<double>(0)) {
		caffe_gpu_add_scalar(n, a, r);
	}
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
		float* r) {
	CURAND_CHECK(
			hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
		double* r) {
	CURAND_CHECK(
			hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

template <typename Dtype>
__global__  void caffe_gpu_csr_gemm_kernel(const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
		const Dtype alpha, int nzz, const Dtype* A, const int* indices,const int* ptr, const Dtype* B, const Dtype beta,
		Dtype* C, const CBLAS_ORDER orderC){

	if (orderC == CblasRowMajor ){
		if (TransB == CblasNoTrans ){
			for( int rowA = blockIdx.x*blockDim.x + threadIdx.x; rowA < M; rowA += blockDim.x * gridDim.x ){
				const int begin = ptr[rowA];
				const int end = ptr[rowA+1];
				for( int colC = blockIdx.y*blockDim.y + threadIdx.y;  colC < N; colC += blockDim.y * gridDim.y ){
					Dtype entry = 0.0;
					for (int pos = begin; pos < end; pos++){
						const int colA = indices[pos];
						entry += A[pos] * B[colA * N + colC];
					}
					const int offsetC = N * rowA + colC;
					C[offsetC] = beta * C[offsetC] + alpha * entry;
				}
			}
		}else{
			for( int rowA = blockIdx.x*blockDim.x + threadIdx.x; rowA < M; rowA += blockDim.x * gridDim.x ){
				const int begin = ptr[rowA];
				const int end = ptr[rowA+1];
				for( int colC = blockIdx.y*blockDim.y + threadIdx.y;  colC < N; colC += blockDim.y * gridDim.y ){
					Dtype entry = 0.0;
					for (int pos = begin; pos < end; pos++){
						const int colA = indices[pos];
						entry += A[pos] * B[colA + colC * K];
					}
					const int offsetC = N * rowA + colC;
					C[offsetC] = beta * C[offsetC] + alpha * entry;
				}
			}
		}
	}else{
		if (TransB == CblasNoTrans ){
			for( int rowA = blockIdx.x*blockDim.x + threadIdx.x; rowA < M; rowA += blockDim.x * gridDim.x ){
				const int begin = ptr[rowA];
				const int end = ptr[rowA+1];
				for( int colC = blockIdx.y*blockDim.y + threadIdx.y;  colC < N; colC += blockDim.y * gridDim.y ){
					Dtype entry = 0.0;
					for (int pos = begin; pos < end; pos++){
						const int colA = indices[pos];
						entry += A[pos] * B[colA * N + colC];
					}
					const int offsetC = rowA + colC * M;
					C[offsetC] = beta * C[offsetC] + alpha * entry;
				}
			}
		}else{
			for( int rowA = blockIdx.x*blockDim.x + threadIdx.x; rowA < M; rowA += blockDim.x * gridDim.x ){
				const int begin = ptr[rowA];
				const int end = ptr[rowA+1];
				for( int colC = blockIdx.y*blockDim.y + threadIdx.y;  colC < N; colC += blockDim.y * gridDim.y ){
					Dtype entry = 0.0;
					for (int pos = begin; pos < end; pos++){
						const int colA = indices[pos];
						entry += A[pos] * B[colA + colC * K];
					}
					const int offsetC = rowA + colC * M;
					C[offsetC] = beta * C[offsetC] + alpha * entry;
				}
			}
		}
	}
}

//C = alpha A * B^T +  C where A and B are vectors. A is a sprase vector and B is a dense vector
template <typename Dtype>
__global__  void caffe_gpu_csr_rank1_update_kernel(const int M, const int N,
		const Dtype alpha, const Dtype* A, const int* indices,const int* ptr, const Dtype* B, int ldb,
		Dtype* C,const CBLAS_ORDER orderC){

	const int begin = ptr[0];
	const int num_entries = ptr[1] - begin;
	if (orderC == CblasRowMajor ){
		for( int pos = blockIdx.x*blockDim.x + threadIdx.x; pos < num_entries; pos += blockDim.x * gridDim.x ){
			const int cor_pos = begin+pos;
			const int rowC = indices[cor_pos];
			const Dtype valA = A[cor_pos];
			for( int colC = blockIdx.y*blockDim.y + threadIdx.y;  colC < N; colC += blockDim.y * gridDim.y ){
				const int C_offset  = rowC * N + colC;
				C[C_offset] += B[colC * ldb] * valA * alpha ;
			}
		}
	}else{
		for( int pos = blockIdx.x*blockDim.x + threadIdx.x; pos < num_entries; pos += blockDim.x * gridDim.x ){
			const int cor_pos = begin+pos;
			const int rowC = indices[cor_pos];
			const Dtype valA = A[cor_pos];
			for( int colC = blockIdx.y*blockDim.y + threadIdx.y;  colC < N; colC += blockDim.y * gridDim.y ){
				const int C_offset  = rowC + colC * M;
				C[C_offset] += B[colC * ldb] * valA * alpha ;
			}
		}
	}
}

template <>
void caffe_gpu_csr_gemm<float>(const CBLAS_TRANSPOSE TransA,
		const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
		const float alpha, int nzz, const float* A, const int* indices, const int* ptr, const float* B, const float beta,
		float* C, const CBLAS_ORDER orderC) {
	std::cout << "getting into new code-------------=========================\n\n\n\n\n";
	if (TransA == CblasNoTrans){
		dim3    grids(CAFFE_GET_2D_BLOCKS(M),CAFFE_GET_2D_BLOCKS(M));
		dim3    threads(CAFFE_GET_2D_THREADS(M), CAFFE_GET_2D_THREADS(N));
		std::cout << "passed grid-------------========================="<< CAFFE_GET_BLOCKS(M) <<", " << CAFFE_GET_BLOCKS(N)<< ", " << CAFFE_CUDA_NUM_THREADS <<"\n\n\n\n\n";
		caffe_gpu_csr_gemm_kernel<float><<<grids,threads>>>(TransB, M, N, K,alpha, nzz, A,  indices, ptr,B, beta, C, orderC);
		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
		    printf("Error: %s\n", hipGetErrorString(err));
		std::cout << "done kernel------------=========================\n\n\n\n\n";
	}else{
		//scale C by beta
		if (beta != 1.0){
			CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle() , M * N, &beta, C, 1));
		}
		dim3 grids(CAFFE_GET_2D_BLOCKS(nzz/K+1),CAFFE_GET_2D_BLOCKS(N));
		dim3 threads(CAFFE_GET_2D_THREADS(nzz/K+1), CAFFE_GET_2D_THREADS(N));
		if (TransB == CblasNoTrans){
			for (int i=0; i < K; i++){
				caffe_gpu_csr_rank1_update_kernel<float><<<grids,threads>>>( M,  N, alpha, A, indices, ptr + i, B+(N*i), 1,C,orderC);
			}
		}else{
			for (int i=0; i < K; i++){
				caffe_gpu_csr_rank1_update_kernel<float><<<grids,threads>>>( M,  N, alpha, A, indices, ptr + i, B+i, K,C,orderC);
			}
		}
	}
}

template <>
void caffe_gpu_csr_gemm<double>(const CBLAS_TRANSPOSE TransA,
		const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
		const double alpha, int nzz, const double* A, const int* indices, const int* ptr, const double* B, const double beta,
		double* C, const CBLAS_ORDER orderC) {
	if (TransA == CblasNoTrans){
		dim3    grids(CAFFE_GET_2D_BLOCKS(M),CAFFE_GET_2D_BLOCKS(M));
		dim3    threads(CAFFE_GET_2D_THREADS(M), CAFFE_GET_2D_THREADS(N));
		caffe_gpu_csr_gemm_kernel<double><<<grids,threads>>>(TransB , M, N, K,alpha, nzz, A,  indices, ptr,B, beta, C, orderC);
	}else{
		//scale C by beta
		if (beta != 1.0){
			CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle() , M * N, &beta, C, 1));
		}
		dim3 grids(CAFFE_GET_2D_BLOCKS(nzz/K+1),CAFFE_GET_2D_BLOCKS(N));
		dim3 threads(CAFFE_GET_2D_THREADS(nzz/K+1), CAFFE_GET_2D_THREADS(N));
		if (TransB == CblasNoTrans){
			for (int i=0; i < K; i++){
				caffe_gpu_csr_rank1_update_kernel<double><<<grids,threads>>>( M,  N, alpha, A, indices, ptr + i, B+(N*i), 1,C,orderC);
			}
		}else{
			for (int i=0; i < K; i++){
				caffe_gpu_csr_rank1_update_kernel<double><<<grids,threads>>>( M,  N, alpha, A, indices, ptr + i, B+i, K,C,orderC);
			}
		}
	}
}





}  // namespace caffe
